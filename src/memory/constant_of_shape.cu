#include "hip/hip_runtime.h"
#include "cudakernel/memory/constant_of_shape.h"
#include "ppl/nn/common/tensor_shape.h"
#include "ppl/common/retcode.h"
#include "ppl/common/types.h"
#include <hip/hip_runtime.h>

template <typename T>
__global__ void ppl_cukernel_constant_of_shape(
    int64_t num_elems,
    const T *pre_set_value,
    T *output)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= num_elems)
        return;

    output[index] = pre_set_value[0];
}

ppl::common::RetCode PPLCUDAConstantOfShapeForwardImp(
    hipStream_t stream,
    const void* pre_set_value,
    const ppl::nn::TensorShape* output_shape,
    void* output)
{
    int64_t num_elems = output_shape->GetElementsIncludingPadding();
    int block_size = 256;
    int grid_size  = (num_elems + block_size - 1) / block_size;

#define SWITCH_CASE(TYPE)                                                      \
    case sizeof(TYPE): {                                                       \
        ppl_cukernel_constant_of_shape<<<grid_size, block_size, 0, stream>>>( \
            num_elems, (const TYPE *)pre_set_value, (TYPE *)output);           \
        return ppl::common::RC_SUCCESS;                                        \
    }

    switch (ppl::common::GetSizeOfDataType(output_shape->GetDataType())) {
        SWITCH_CASE(int8_t);
        SWITCH_CASE(int16_t);
        SWITCH_CASE(int32_t);
        SWITCH_CASE(int64_t);
        default:
            return ppl::common::RC_UNSUPPORTED;
    }
#undef SWITCH_CASE
}