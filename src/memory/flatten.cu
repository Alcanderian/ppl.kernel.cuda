#include "cudakernel/memory/flatten.h"
#include "ppl/nn/common/tensor_shape.h"
#include "ppl/common/retcode.h"
#include "ppl/common/types.h"
#include <hip/hip_runtime.h>

ppl::common::RetCode PPLCUDAFlattenForwardImp(
    hipStream_t stream,
    const ppl::nn::TensorShape* input_shape,
    const void* input,
    const ppl::nn::TensorShape* output_shape,
    void* output)
{
    int64_t num_elems_output = output_shape->GetElementsIncludingPadding();
    hipMemcpyAsync(output, input, ppl::common::GetSizeOfDataType(input_shape->GetDataType()) * num_elems_output, hipMemcpyDeviceToDevice, stream);
    return ppl::common::RC_SUCCESS;
}